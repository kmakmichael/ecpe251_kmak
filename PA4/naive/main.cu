/*
    Michael Kmak    
    ECPE 251 - High-Performance Computing
    PA4 - Canny Edge Sate 1 - GPU

    usage: ./canny <image path> <sigma>

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

//#include "sort.h"
#include "image_template.h"

#define GPU_NO 1 // 85 % 4

int main(int argc, char *argv[]) {

    float sigma;
    if (argc != 3) {
        fprintf(stderr, "usage: ./canny <image path> <sigma>\n");
        return -1;
    }
    sigma = atof(argv[2]);
    if (sigma <= 0) {
        fprintf(stderr, "invalid sigma: %s\n", argv[2]);
        return -1;
    }

    hipSetDevice(GPU_NO);
}
