#include "hip/hip_runtime.h"
/*
    Michael Kmak    
    ECPE 251 - High-Performance Computing
    PA4 - Canny Edge Sate 1 - GPU

    usage: ./canny <image path> <sigma>

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

//#include "sort.h"
#include "image_template.h"

#define GPU_NO 1 // 85 % 4
#define BLOCKSIZE 16

void print_k(float *k, int len);
void g_kern(float *k, float sigma);
void g_deriv(float *k, float sigma);


int main(int argc, char *argv[]) {

    int height = 0;
    int width = 0;
    float sigma = 0.0;
    int kern_w = 0;

    // host
    float *h_img;
    float *h_mag;
    float *h_dir;
    float *h_vkern;
    float *h_hkern;
    float *h_vderiv;
    float *h_hderiv;

    // device
    float *d_img;
    float *d_temp;
    float *d_hori;
    float *d_vert;
    float *d_mag;
    float *d_dir;
    float *d_vkern;
    float *d_hkern;
    float *d_vderiv;
    float *d_hderiv;
 
    // argparse
    if (argc != 3) {
        fprintf(stderr, "usage: ./canny <image path> <sigma>\n");
        return -1;
    }
    sigma = atof(argv[2]);
    if (sigma <= 0) {
        fprintf(stderr, "invalid sigma: %s\n", argv[2]);
        return -1;
    }

    // cuda setup
    hipSetDevice(GPU_NO);
    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 dimGrid(width/BLOCKSIZE, height/BLOCKSIZE);

    // image prep
    read_image_template(argv[1], &h_img, &width, &height);
    h_mag = (float *) calloc(width*height, sizeof(float));
    h_dir = (float *) calloc(width*height, sizeof(float));
    hipMalloc((void **)&d_img, sizeof(float)*width*height);
    hipMalloc((void **)&d_temp, sizeof(float)*width*height);
    hipMalloc((void **)&d_hori, sizeof(float)*width*height);
    hipMalloc((void **)&d_vert, sizeof(float)*width*height);
    hipMalloc((void **)&d_mag, sizeof(float)*width*height);
    hipMalloc((void **)&d_dir, sizeof(float)*width*height);

    // prepare canny kernels
    kern_w = 2 * round(2.5 * sigma - 0.5) + 1;
    h_vkern = (float *) calloc(kern_w, sizeof(float));
    h_hkern = (float *) calloc(kern_w, sizeof(float));
    h_vderiv = (float *) calloc(kern_w, sizeof(float));
    h_hderiv = (float *) calloc(kern_w, sizeof(float));
    hipMalloc((void **)&d_vkern, sizeof(float)*kern_w);
    hipMalloc((void **)&d_hkern, sizeof(float)*kern_w);
    hipMalloc((void **)&d_vderiv, sizeof(float)*kern_w);
    hipMalloc((void **)&d_hderiv, sizeof(float)*kern_w);

    // calculate ckernels
    g_kern(h_vkern, sigma);
    g_kern(h_hkern, sigma);
    g_deriv(h_vderiv, sigma);
    g_deriv(h_hderiv, sigma);

    // transfer ckernels
    hipMemcpy(d_vkern, h_vkern, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_hkern, h_hkern, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_vderiv, h_vderiv, sizeof(float)*kern_w, hipMemcpyHostToDevice);
    hipMemcpy(d_hderiv, h_hderiv, sizeof(float)*kern_w, hipMemcpyHostToDevice);

    // GPU convolution
        


    // free
    free(h_vkern);
    free(h_hkern);
    free(h_vderiv);
    free(h_hderiv);
    free(h_mag);
    free(h_dir);
    hipFree(d_img);
    hipFree(d_temp);
    hipFree(d_hori);
    hipFree(d_vert);
    hipFree(d_mag);
    hipFree(d_dir);
}


void print_k(float *k, int len) {
    for (size_t i = 0; i < len; i++) {
        printf("[%f]", k[i]);
    }
    printf("\n");
}


void g_kern(float *k, float sigma) {
    float a = round(2.5 * sigma - 0.5);
    int w = 2 * a + 1;
    float sum = 0;

    for (size_t i = 0; i < w; i++) {
        k[i] = exp((-1.0 * (i-a) * (i-a)) / (2.0 * sigma * sigma));
        sum += k[i];
    }
    for (size_t i = 0; i < w; i++) {
        k[i] /= sum;
    }
}


void g_deriv(float *k, float sigma) {
    float a = round(2.5 * sigma - 0.5);
    int w = 2 * a + 1;
    float sum = 0;
    
    for (size_t i = 0; i < w; i++) {
        k[i] = -1.0 * (i-a) * exp((-1.0 * (i-a) * (i-a)) / (2.0 * sigma * sigma));
        sum -= i * k[i];
    }
    for (size_t i = 0; i < w; i++) {
        k[i] /= sum;
    }
    // flip
    for (size_t i = 0; i < (w/2); i++) {
        float temp = k[w-1-i];
        k[w-1-i] = k[i];
        k[i] = temp;
    }
}
