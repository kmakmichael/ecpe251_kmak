/*
    Michael Kmak
    ECPE 251 - High-Performance Computing
    PA6 - Performance Prediction

    usage: ./memcpy size

    copies a randomly-filled float array between
    the CPU and GPU in both directions. size of
    this array is 2^N bytes, where N is given as
    a program argument. outputs time taken in 
    the following format:
        size H2D D2H
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define GPU_NO 1 // 85 % 4


int main(int argc, char *argv[]) {
    
    float *h_arr;
    float *d_arr;
    size_t n;

    // argparse
    if (argc != 2) {
        fprintf(stderr, "usage: ./memcpy <size>\n");
        return 1;
    }
    int success = sscanf(argv[1], "%zu", &n);
    if (success != 1) {
        fprintf(stderr, "invalid size, enter an integer\n");
        return -1;
    }
    if (n > log2((float)SIZE_MAX)) {
        fprintf(stderr, "enter a power of two no larger than %f\n", log2((float)SIZE_MAX));
        return -1;
    }

    printf("Creating array of floats. size 2^%zu=%f bytes, containing %f floats\n", n, exp2((float)n), exp2((float)n)/sizeof(float));
    n = exp2((float)n);
}
